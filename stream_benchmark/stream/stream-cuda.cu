#include "hip/hip_runtime.h"
#include <chrono>

#include "../util.h"
#include "stream-util.h"

__global__ void stream(size_t nx, const double *__restrict__ src, double *__restrict__ dest) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<nx)
        dest[i] = src[i] + 1;
}

int main(int argc, char *argv[]) {
    size_t nx, nItWarmUp, nIt;
    parseCLA_1d(argc, argv, nx, nItWarmUp, nIt);

    //allocate memory

    size_t size = sizeof(double) * nx;

    double *src, *dest;
    hipHostMalloc(&src, size);
    hipHostMalloc(&dest, size);
    
    double *d_src, *d_dest;
    hipMalloc(&d_src, size);
    hipMalloc(&d_dest, size);

    
    // init
    initStream(src, nx);
    // copy from cpu to gpu
    hipMemcpy(d_src, src, size, hipMemcpyHostToDevice);

    auto numThreadsPerBlock = 64 ;
    auto numBlocks = (nx+ numThreadsPerBlock-1) / numThreadsPerBlock;

    // warm-up
    for (int i = 0; i < nItWarmUp; ++i) {
        stream<<<numBlocks, numThreadsPerBlock>>>(nx, d_src, d_dest);
        hipDeviceSynchronize();
        std::swap(d_src, d_dest);
    }

    // measurement
    auto start = std::chrono::steady_clock::now();

    for (int i = 0; i < nIt; ++i) {
        stream<<<numBlocks, numThreadsPerBlock>>>(nx, d_src, d_dest);
        hipDeviceSynchronize();
        std::swap(d_src, d_dest);
    }

    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();

    //copy from GPU to CPU --> we swapped d_dest with d_src.. thats why we use d_src here
    hipMemcpy(dest, d_src, size, hipMemcpyDeviceToHost);

    printStats(end - start, nx, nIt, streamNumReads, streamNumWrites);

    // check solution --> the result is in dest
    checkSolutionStream(dest, nx, nIt + nItWarmUp);

    hipFree(d_src);
    hipFree(d_dest);

    hipHostFree(src);
    hipHostFree(dest);

    return 0;
}
